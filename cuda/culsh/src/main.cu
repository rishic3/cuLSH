#include "rplsh/index.cuh"
#include "rplsh/kernels/build_index.cuh"
#include "rplsh/kernels/hash.cuh"
#include "rplsh/kernels/projections.cuh"
#include "rplsh/kernels/query_index.cuh"

#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iomanip>
#include <iostream>
#include <vector>

struct Params {
    int n_hash_tables;
    int n_projections;
    int seed;
};

void generate_x(float* X, int n, int d, int seed) {
    hiprandGenerator_t rng;
    CURAND_CHECK(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(rng, seed));

    size_t x_size = static_cast<size_t>(n) * d;
    CURAND_CHECK(hiprandGenerateUniform(rng, X, x_size));

    CURAND_CHECK(hiprandDestroyGenerator(rng));
}

culsh::rplsh::Index main_fit(hipblasHandle_t cublas_handle, hipStream_t stream, const float* X,
                             int n_samples, int n_features, const Params& params, float* P) {

    // allocate X_hash
    float* X_hash;
    CUDA_CHECK(hipMalloc(&X_hash, static_cast<size_t>(n_samples) * params.n_hash_tables *
                                       params.n_projections * sizeof(float)));

    // generate random projections and hash X
    culsh::rplsh::detail::generate_random_projections<float>(
        stream, params.n_hash_tables * params.n_projections, n_features, params.seed, P);
    culsh::rplsh::detail::hash<float>(cublas_handle, stream, X, P, n_samples, n_features,
                                      params.n_hash_tables, params.n_projections, X_hash);

    // compute binary signatures from X_hash
    int8_t* X_sig;
    CUDA_CHECK(hipMalloc(&X_sig, static_cast<size_t>(n_samples) * params.n_hash_tables *
                                      params.n_projections * sizeof(int8_t)));
    culsh::rplsh::detail::compute_signatures<float>(stream, X_hash, n_samples, params.n_hash_tables,
                                                    params.n_projections, X_sig);
    CUDA_CHECK(hipFree(X_hash)); // done with X_hash

    // build and return index
    auto index = culsh::rplsh::detail::build_index(stream, X_sig, n_samples, params.n_hash_tables,
                                                   params.n_projections);
    CUDA_CHECK(hipFree(X_sig));

    return index;
}

void test() {
    const int n = 10000000;
    const int d = 512;
    const int n_hash_tables = 64;
    const int n_projections = 8;
    const int n_total_buckets = n_hash_tables * n_projections;

    // create stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // create cublas handle
    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));

    float* X;
    float* P;

    // allocate memory for X, P, X_hash
    CUDA_CHECK(hipMalloc(&X, static_cast<size_t>(n) * d * sizeof(float)));
    CUDA_CHECK(hipMalloc(&P, static_cast<size_t>(n_total_buckets) * d * sizeof(float)));

    generate_x(X, n, d, 12345);

    auto params = Params{n_hash_tables, n_projections, 12345};

    auto start_fit = std::chrono::high_resolution_clock::now();
    // Fit the LSH model and get the index
    culsh::rplsh::Index index = main_fit(cublas_handle, stream, X, n, d, params, P);
    auto end_fit = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> fit_time = end_fit - start_fit;

    std::cout << "Built index with " << index.n_hash_tables << " hash tables and "
              << index.n_projections << " projections per table" << std::endl;
    std::cout << "  Fit took [" << fit_time.count() << "s]" << std::endl;

    CUDA_CHECK(hipFree(X));

    // QUERY
    float* Q;
    CUDA_CHECK(hipMalloc(&Q, static_cast<size_t>(n) * d * sizeof(float)));
    generate_x(Q, n, d, 42);

    float* Q_hash;
    CUDA_CHECK(hipMalloc(&Q_hash, static_cast<size_t>(n) * n_total_buckets * sizeof(float)));

    auto start_query = std::chrono::high_resolution_clock::now();

    culsh::rplsh::detail::hash<float>(cublas_handle, stream, Q, P, n, d, n_hash_tables,
                                      n_projections, Q_hash);

    CUDA_CHECK(hipFree(P));

    int8_t* Q_sig;
    CUDA_CHECK(hipMalloc(&Q_sig, static_cast<size_t>(n) * n_total_buckets * sizeof(int8_t)));

    // convert hash values to signatures
    culsh::rplsh::detail::compute_signatures<float>(stream, Q_hash, n, n_hash_tables, n_projections,
                                                    Q_sig);

    CUDA_CHECK(hipFree(Q_hash));

    culsh::rplsh::Candidates candidates =
        culsh::rplsh::detail::query_index(stream, Q_sig, n, n_hash_tables, n_projections, &index);

    auto end_query = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> query_time = end_query - start_query;
    std::cout << "  Query took [" << query_time.count() << "s]" << std::endl;

    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    CUDA_CHECK(hipStreamDestroy(stream));
}

void test_breakdown() {
    const int n = 1000000;
    const int d = 128;
    const int n_hash_tables = 64;
    const int n_projections = 8;
    const int n_total_buckets = n_hash_tables * n_projections;

    float* X;
    float* P;
    float* X_hash;

    // create stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // create cublas handle
    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));

    // allocate memory for X, P, X_hash
    CUDA_CHECK(hipMalloc(&X, static_cast<size_t>(n) * d * sizeof(float)));
    CUDA_CHECK(hipMalloc(&P, static_cast<size_t>(n_total_buckets) * d * sizeof(float)));
    CUDA_CHECK(hipMalloc(&X_hash, static_cast<size_t>(n) * n_total_buckets * sizeof(float)));

    // generate X
    generate_x(X, n, d, 12345);

    auto start_generate_projections = std::chrono::high_resolution_clock::now();
    // generate random projections
    culsh::rplsh::detail::generate_random_projections<float>(stream, n_total_buckets, d, 12345, P);
    CUDA_CHECK(hipStreamSynchronize(stream));
    auto end_generate_projections = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> generate_projections_time =
        end_generate_projections - start_generate_projections;
    std::cout << "Generated projections in " << generate_projections_time.count() << " sec"
              << std::endl;

    auto start_hash = std::chrono::high_resolution_clock::now();
    // hash
    culsh::rplsh::detail::hash<float>(cublas_handle, stream, X, P, n, d, n_hash_tables,
                                      n_projections, X_hash);
    CUDA_CHECK(hipStreamSynchronize(stream));
    auto end_hash = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> hash_time = end_hash - start_hash;
    std::cout << "Hashed in " << hash_time.count() << " sec" << std::endl;

    // free input and projections
    CUDA_CHECK(hipFree(P));
    CUDA_CHECK(hipFree(X));

    // allocate memory for signatures
    int8_t* X_signatures;
    CUDA_CHECK(
        hipMalloc(&X_signatures, static_cast<size_t>(n) * n_total_buckets * sizeof(int8_t)));

    auto start_compute_signatures = std::chrono::high_resolution_clock::now();
    // convert hash values to signatures
    culsh::rplsh::detail::compute_signatures<float>(stream, X_hash, n, n_hash_tables, n_projections,
                                                    X_signatures);
    // CUDA_CHECK(hipStreamSynchronize(stream));
    auto end_compute_signatures = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> compute_signatures_time =
        end_compute_signatures - start_compute_signatures;
    std::cout << "Computed signatures in " << compute_signatures_time.count() << " sec"
              << std::endl;

    auto start_build_index = std::chrono::high_resolution_clock::now();
    // build index
    culsh::rplsh::Index index =
        culsh::rplsh::detail::build_index(stream, X_signatures, n, n_hash_tables, n_projections);
    // CUDA_CHECK(hipStreamSynchronize(stream));
    auto end_build_index = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> build_index_time = end_build_index - start_build_index;
    std::cout << "Built index in " << build_index_time.count() << " sec" << std::endl;

    std::chrono::duration<double> total_time = end_build_index - start_generate_projections;
    std::cout << "   Total fit time in " << total_time.count() << " sec" << std::endl;
    // print index metadata
    std::cout << "Index metadata: " << std::endl;
    std::cout << "  n_total_buckets: " << index.n_total_buckets << std::endl;
    std::cout << "  n_hash_tables: " << index.n_hash_tables << std::endl;
    std::cout << "  n_projections: " << index.n_projections << std::endl;

    CUDA_CHECK(hipGetLastError());

    // free signatures
    CUDA_CHECK(hipFree(X_signatures));
    CUDA_CHECK(hipFree(X_hash));
    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    CUDA_CHECK(hipStreamDestroy(stream));
}

int main() {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    std::cout << "Using GPU: " << prop.name << std::endl;
    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Global Memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB\n" << std::endl;

    // test_generate_random_projections(false);
    // test_hash();
    test();
    return 0;
}
